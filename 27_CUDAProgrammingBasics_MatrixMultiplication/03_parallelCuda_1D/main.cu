/*===========================================================================*\
  * matrix multiplication code on cuda

  * reference for error checking code snippet
https://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api

  * Ramkumar
  * Thu Feb  6 14:39:46 IST 2025
\*===========================================================================*/

// preprocessor directives

#include <hip/hip_runtime.h>
#include<iostream>
#include<chrono>
#include<random>
#include<fstream>

// defining size of square matrix
const int N = 4001;

// function declarations
void printMatrix(int *A);
__global__ void matMul(int *A, int *B, int *C);
__global__ void hadamardProduct(int *A, int *B);
void initMat(int *A);
void writeMatrix(int *A, std::string name);

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

/*---------------------------------------------------------------------------*/

int main(int argc, char* argv[]){

    // defining 2D pointer (double pointers)
    int *A,*B,*C;
    A = new int [N*N];
    B = new int [N*N];
    C = new int [N*N];

    initMat(A);
    writeMatrix(A,"A.mat");
    initMat(B);
    writeMatrix(B,"B.mat");

    // allocating memory on device
    int *d_A,*d_B,*d_C;

    gpuErrchk( hipMalloc((void**)&d_A, N*N*sizeof(int*)) );
    gpuErrchk( hipMalloc((void**)&d_B, N*N*sizeof(int*)) );
    gpuErrchk( hipMalloc((void**)&d_C, N*N*sizeof(int*)) );

    // copying host **array to device
    gpuErrchk( hipMemcpy(d_A,A,N*N*sizeof(int),hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(d_B,B,N*N*sizeof(int),hipMemcpyHostToDevice) );

    // defining number of threads
    dim3 noOfThreads(32,32);
    dim3 noOfBlocks(126,126);

    // setting timer
    auto timerStart = std::chrono::high_resolution_clock::now();
    matMul<<<noOfBlocks,noOfThreads>>>(d_A,d_B,d_C);
    gpuErrchk( hipPeekAtLastError() );
    hipDeviceSynchronize();
    auto timerStop = std::chrono::high_resolution_clock::now();


    // getting back the result matrix
    hipMemcpy(C,d_C, N*N*sizeof(int), hipMemcpyDeviceToHost);

    auto duration = std::chrono::duration_cast<std::chrono::seconds>(timerStop-timerStart);


    std::cout << std::endl;

    writeMatrix(C,"C.mat");

    std::cout << "Time elapsed : " << duration.count() << std::endl;

    return 0;
}

// function definitions

void printMatrix(int *A){
    for(int i=0; i<N; i++){ // rows
        for(int j=0; j<N; j++){ // columns
            int idx = j+i*N; // c+r*N
            std::cout << " " << A[idx];
        }
        std::cout << std::endl;
    }
}

__global__
void hadamardProduct(int *A, int *B){
    int i = threadIdx.x+blockDim.x*blockIdx.x;
    int j = threadIdx.y+blockDim.y*blockIdx.y;
    int idx = j+i*N;
    B[idx] = A[idx]*A[idx];
}

__global__
void matMul(int *A, int *B, int *C){
    int i = threadIdx.x+blockDim.x*blockIdx.x;
    int j = threadIdx.y+blockDim.y*blockIdx.y;
    if (i<N && j<N){
        int idx = j+i*N;
        for(int k=0; k<N; k++)
            C[idx] += A[k+i*N]*B[j+k*N];
    }
}

void initMat(int *A){
    for(int i=0; i<N; i++){ // rows
        for(int j=0; j<N; j++){ // columns
            int idx = j+i*N;
            A[idx] = random()%9+1;
        }
    }
}

void writeMatrix(int *A, std::string name){
    std::ofstream fid(name);
    for(int i=0; i<N; i++){ // rows
        for(int j=0; j<N; j++){ // columns
            int idx = j+i*N;
            fid << A[idx] << ",";
        }
        fid << std::endl;
    }
    fid.close();
}


/*---------------------------------------------------------------------------*/
